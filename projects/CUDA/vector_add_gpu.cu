
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel to add elements of two arrays
__global__ void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void) {

    // Create events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // int N = 1<<20; // 1M elements
    int64_t N = 1LL<<30; // 1 billion elements

    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    hipError_t cudaStatus;
    cudaStatus = hipMallocManaged(&x, N*sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocManaged failed!");
        return 1;
    }

    cudaStatus = hipMallocManaged(&y, N*sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocManaged failed!");
        hipFree(x);
        return 1;
    }

    // Initialize x and y arrays on the host
    for (int64_t i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Record the start event
    hipEventRecord(start, NULL);

    // Run kernel on 1M elements on the GPU
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);


    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching add kernel!", cudaStatus);
        hipFree(x);
        hipFree(y);
        return 1;
    }

    // Record the stop event
    hipEventRecord(stop, NULL);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("GPU time taken: %f milliseconds\n", milliseconds);

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    printf("Max error: %f\n", maxError);

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}
